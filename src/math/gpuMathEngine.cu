#include "hip/hip_runtime.h"

#include <functional>
#include <iostream>
#include <cmath>

// GLM - Cuda interoperability
#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>

#include "..\..\include\math\gpuMathEngine.hpp"

#define STREAMS 12
#define WARPS2D 4
#define WARPS WARPS2D*WARPS2D
#define M_PI 3.14159265358979323846

// DEVICE SETUP

GPUMathEngine::GPUMathEngine()
{
    cudaCheck( hipDeviceGetAttribute(&nSM, hipDeviceAttributeMultiprocessorCount, 0) );
}

// KERNELS

__device__
int k_combine(int x, int y) {
    return (x*12345) + y;
}

__device__
float k_hashFunction(int x)
{
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    return ( x % 101 ) / 100.0f;
}

__device__
int k_centHash(int x)
{
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    return ( x % 201 ) - 100;
}

__device__
float k_lerp(float a, float b, float x)
{
    return a + x * (b - a);
}

__device__
float k_fade(float x)
{
    return x * x * x * (x * (x * 6 - 15) + 10);
}

__global__
void perlinSample(float *out, int dimension, float min, float max, float period)
{
    
    // Start index
    int startIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Stride
    int index = startIndex;
    int x, y;
    do
    {
        // Thread calculation
        x = index % dimension;
        y = index / dimension;

        // Square index
        int X = std::floor( x / period );
        int Y = std::floor( y / period );

        // Normal relative position
        float rx = (x/period) - X;
        float ry = (y/period) - Y;

        // Square corner vectors
        glm::vec2 BL = glm::normalize( glm::vec2( k_centHash( k_combine( X , Y ) ), k_centHash( k_combine( X , Y )+1 ) ) );
        glm::vec2 BR = glm::normalize( glm::vec2( k_centHash( k_combine(X+1, Y ) ), k_centHash( k_combine(X+1, Y )+1 ) ) );
        glm::vec2 TL = glm::normalize( glm::vec2( k_centHash( k_combine( X ,Y+1) ), k_centHash( k_combine( X ,Y+1)+1 ) ) );
        glm::vec2 TR = glm::normalize( glm::vec2( k_centHash( k_combine(X+1,Y+1) ), k_centHash( k_combine(X+1,Y+1)+1 ) ) );

        // Relational vectors
        glm::vec2 point = glm::vec2( rx, ry );
        glm::vec2 BLr = glm::vec2( 0, 0 ) - point;
        glm::vec2 BRr = glm::vec2( 1, 0 ) - point;
        glm::vec2 TLr = glm::vec2( 0, 1 ) - point;
        glm::vec2 TRr = glm::vec2( 1, 1 ) - point;

        // Dot products
        float BLd = glm::dot( BL, BLr );
        float BRd = glm::dot( BR, BRr );
        float TLd = glm::dot( TL, TLr );
        float TRd = glm::dot( TR, TRr );

        // Interpolate
        float bottom = k_lerp( BLd, BRd, k_fade(point.x) );
        float top = k_lerp( TLd, TRd, k_fade(point.x) );
        float centre = k_lerp( bottom, top, k_fade(point.y) );

        // Set value
        out[index] = ( ((centre+1) / 2) * (max-min) ) + min;

        // Stride
        index += blockDim.x*gridDim.x;
    }
    while ( y<dimension );
}

__global__
void hashSample(float *out, int dimension, float min, float max, float period)
{
    
    // Start index
    int startIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Stride
    int index = startIndex;
    int y;
    do
    {
        y = index / dimension;

        // Set value
        out[index] = min + ( k_hashFunction(index) * (max-min) );

        // Stride
        index += blockDim.x*gridDim.x;
    }
    while ( y<dimension );
}

// FUNCTIONS

void GPUMathEngine::generateHeightMap(int dimension, float min, float max, float *out, Sample sample, float period, int octaves)
{
    // Allocate device memory
    float *d_out;
    cudaCheck( hipMalloc( (void **)&d_out, dimension*dimension*sizeof(float) ) );
    switch ( sample )
    {
    case hash:
        hashSample<<<nSM, WARPS*32>>>(d_out, dimension, min, max, period);
        break;
    case perlin:
        perlinSample<<<nSM, WARPS*32>>>(d_out, dimension, min, max, period);
        break;
    default:
        hashSample<<<nSM, WARPS*32>>>(d_out, dimension, min, max, period);
        break;
    }
    cudaCheck( hipMemcpy(out, d_out, dimension*dimension*sizeof(float), hipMemcpyDeviceToHost) );
    cudaCheck( hipFree(d_out) );
}

// MACROS

inline void GPUMathEngine::cudaCheck(hipError_t err)
{
    if (err != hipSuccess)
        std::cout << "Cuda error: " << hipGetErrorString(err) << std::endl;
}

inline void GPUMathEngine::multiCudaMalloc(int size, void **a, void **b, void **c)
{
    cudaCheck( hipMalloc(a, size) );
    if (b != NULL) cudaCheck( hipMalloc(b, size) );
    if (c != NULL) cudaCheck( hipMalloc(c, size) );
}

inline void GPUMathEngine::multiCudaFree(void *a, void *b, void *c)
{
    cudaCheck( hipFree(a) );
    if (b != NULL) cudaCheck( hipFree(b) );
    if (c != NULL) cudaCheck( hipFree(c) );
}
