#include "hip/hip_runtime.h"

#include "..\..\include\math\cpuMathEngine.hpp"

// GLM
#include <glm/glm.hpp>

// Host code
namespace cpucommon
{
    #include "..\..\src\math\common.cu"
}

// FUNCTIONS

void CPUMathEngine::generateHeightMap(float *out, int dimension, float min, float max, Sample sample, float period, int octaves)
{
    for (int y=0; y<dimension; y++) for (int x=0; x<dimension; x++)
    {
        float value;
        // Custom sampling
        switch ( sample )
        {
        case mountain:
            value = cpucommon::mountain(x, y, period);
            break;
        default:
            value = cpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        out[y*dimension + x] = min + ( value * (max-min) );
    }
}
