#include "hip/hip_runtime.h"

#include <iostream>
#include "..\include\math.hpp"

namespace cudamath {

    __global__
    void strideAdd(int *a, int *b, int *c, int size)
    {
        int index = threadIdx.x + blockIdx.x * blockDim.x;
        for (int i=index; i<size; i += blockDim.x*gridDim.x)
            c[i] = a[i] + b[i];
    }

    void gpuAdd(int *a, int *b, int *c, int n, int warps)
    {
        // Allocate device memory
        int *d_a, *d_b, *d_c;
        cudaCheck( hipMalloc(&d_a, n*sizeof(int)) );
        cudaCheck( hipMalloc(&d_b, n*sizeof(int)) );
        cudaCheck( hipMalloc(&d_c, n*sizeof(int)) );

        // Send memory to device
        cudaCheck( hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice) );
        cudaCheck( hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice) );

        // Run kernel
        int sms;
        cudaCheck( hipDeviceGetAttribute(&sms, hipDeviceAttributeMultiprocessorCount, 0) );
        strideAdd<<<sms, warps*32>>>(d_a, d_b, d_c, n);

        // Fetch memory from device
        cudaCheck( hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost) );

        // Cleanup
        cudaCheck( hipFree(d_a) );
        cudaCheck( hipFree(d_b) );
        cudaCheck( hipFree(d_c) );
    }

    void cpuAdd(int *a, int *b, int *c, int n)
    {
        for (int i=0; i<n; i++)
            c[i] = a[i] + b[i];
    }

    void cudaCheck(hipError_t err)
    {
        if (err != hipSuccess)
            std::cout << "Cuda error: " << hipGetErrorString(err);
    }

}
