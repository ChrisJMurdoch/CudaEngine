#include "hip/hip_runtime.h"

// This has to be directly included into a translation unit as it contains
// device code, wrap include statement in a namespace to avoid linker errors.

__host__ __device__
float floatHash(int x)
{
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    return ( x % 10000 ) / 9999.0f;
}

__host__ __device__
int combine(int x, int y) {
    return (x*12345) + y;
}

__host__ __device__
float lerp(float a, float b, float x)
{
    return a + x * (b - a);
}

__host__ __device__
float fade(float x)
{
    return x * x * x * (x * (x * 6 - 15) + 10);
}

__host__ __device__
float falloff(float x)
{
    const float PI = 3.14159265358979323846;
    return powf( sin(x*PI), 0.05 );
}

// SAMPLES (X,Y,P) => Z

__host__ __device__
float hashSample(int x, int y, float period)
{
    return floatHash( combine(x, y) );
}

__host__ __device__
float sinSample(int x, int y, float period)
{
    const float PI = 3.14159265358979323846;
    float xd = ( sin( x * (2*PI) / period ) + 1 ) / 2;
    float yd = ( sin( y * (2*PI) / period ) + 1 ) / 2;
    return xd * yd;
}

__host__ __device__
float perlinSample(int x, int y, float period)
{
    // Square coords
    int X = std::floor( x / period );
    int Y = std::floor( y / period );

    // Relative point coords
    float rx = (x/period) - X;
    float ry = (y/period) - Y;

    // Square corner vectors
    glm::vec2 BL = glm::normalize( glm::vec2( floatHash( combine( X , Y ) )-0.5, floatHash( combine( X , Y )+1 )-0.5 ) );
    glm::vec2 BR = glm::normalize( glm::vec2( floatHash( combine(X+1, Y ) )-0.5, floatHash( combine(X+1, Y )+1 )-0.5 ) );
    glm::vec2 TL = glm::normalize( glm::vec2( floatHash( combine( X ,Y+1) )-0.5, floatHash( combine( X ,Y+1)+1 )-0.5 ) );
    glm::vec2 TR = glm::normalize( glm::vec2( floatHash( combine(X+1,Y+1) )-0.5, floatHash( combine(X+1,Y+1)+1 )-0.5 ) );

    // Relational vectors
    glm::vec2 point = glm::vec2( rx, ry );
    glm::vec2 BLr = glm::vec2( 0, 0 ) - point;
    glm::vec2 BRr = glm::vec2( 1, 0 ) - point;
    glm::vec2 TLr = glm::vec2( 0, 1 ) - point;
    glm::vec2 TRr = glm::vec2( 1, 1 ) - point;

    // Dot products
    float BLd = glm::dot( BL, BLr );
    float BRd = glm::dot( BR, BRr );
    float TLd = glm::dot( TL, TLr );
    float TRd = glm::dot( TR, TRr );

    // Interpolate using fade
    float bottom = lerp( BLd, BRd, fade(point.x) );
    float top = lerp( TLd, TRd, fade(point.x) );
    float centre = lerp( bottom, top, fade(point.y) );

    // 0-1
    return (centre+1) / 2;
}
