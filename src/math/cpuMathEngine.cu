#include "hip/hip_runtime.h"

#include "..\..\include\math\cpuMathEngine.hpp"

#include "..\..\include\logger\log.hpp"

// GLM
#include <glm/glm.hpp>

// Host code
namespace cpucommon
{
    #include "..\..\src\math\common.cu"
}

#include <cmath>

// FUNCTIONS

void CPUMathEngine::generateHeightMap(float *out, int dimension, float min, float max, Sample sample, float period, int octaves)
{
    for (int y=0; y<dimension; y++) for (int x=0; x<dimension; x++)
    {
        float value;
        // Custom sampling
        switch ( sample )
        {
        case mountain:
            value = cpucommon::mountain(x, y, period);
            break;
        default:
            value = cpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        out[y*dimension + x] = min + ( value * (max-min) );
    }
}

void brush(float *map, int width, int x, int y, float amount, int radius)
{
    int dim = (radius*2)+1;
    amount /= (float)dim*dim;

    // Calc distribution
    float total = 0;
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;
        
        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        total += close;
    }
    float mult = dim*dim / total;

    // Add values
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;

        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        // Paint
        map[ x+xo + ((y+yo)*width) ] += mult*amount*close;
        total++;
    }
}

float getCellHeight(float *map, int width, int x, int y)
{
    return ( x<0 || x>=width || y<0 || y>=width ) ? 9999 : map[ x + (y*width) ]+1000;
}

void erodeCell(float *map, int width, int x, int y, float speed, float sediment, int radius)
{
    // Calc height
    float height = getCellHeight(map, width, x, y);

    // Find lowest cell in 3x3
    int lx=x, ly=y;
    float lh = height;
    for (int xo=-1; xo<2; xo++) for (int yo=-1; yo<2; yo++)
    {
        float h = getCellHeight(map, width, x+xo, y+yo);
        if ( h<lh )
        {
            lx = x+xo;
            ly = y+yo;
            lh = h;
        }
    }

    // Calculate difference
    float delta = height - lh;

    // Stuck
    if ( -delta >= speed )
    {
        brush(map, width, x, y, sediment, radius);
        return;
    }

    // CUSTOM
    speed += delta-0.1f;
    float capacity = speed;
    
    // Alter sediment
    float deposit = sediment - capacity;
    brush(map, width, x, y, deposit, radius);

    erodeCell(map, width, lx, ly, speed, capacity, radius);
}

void CPUMathEngine::erode(float *map, int width, int droplets, int radius)
{
    static float s_i = 0;
    srand( s_i++ );
    for (int i=0; i<droplets; i++)
    {
        int x = rand() % width, y = rand() % width;
        erodeCell(map, width, x, y, 0, 0, radius);
    }
}
