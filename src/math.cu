
#include <iostream>

#include "..\include\math.hpp"

#include ".\kernels.cu"

#define WARPS 8

// VECTOR ADDITION

void cudamath::vectorAdd(int *a, int *b, int *c, int n)
{
    // Host memory -> Device memory
    int *d_a, *d_b, *d_c;
    multiCudaMalloc(n*sizeof(int), (void **)&d_a, (void **)&d_b, (void **)&d_c);
    cudaCheck( hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice) );
    // Run kernel
    int sm; cudaCheck( hipDeviceGetAttribute(&sm, hipDeviceAttributeMultiprocessorCount, 0) );
    kernels::vectorAdd<<<sm, WARPS*32>>>(d_a, d_b, d_c, n);
    // Device memory -> Host memory
    cudaCheck( hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost) );
    multiCudaFree(d_a, d_b, d_c);
}

void cudamath::vectorInAdd(int *a, int *b, int n)
{
    // Host memory -> Device memory
    int *d_a, *d_b;
    multiCudaMalloc(n*sizeof(int), (void **)&d_a, (void **)&d_b);
    cudaCheck( hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice) );
    // Run kernel
    int sm; cudaCheck( hipDeviceGetAttribute(&sm, hipDeviceAttributeMultiprocessorCount, 0) );
    kernels::vectorInAdd<<<sm, WARPS*32>>>(d_a, d_b, n);
    // Device memory -> Host memory
    cudaCheck( hipMemcpy(a, d_a, n*sizeof(int), hipMemcpyDeviceToHost) );
    multiCudaFree(d_a, d_b);
}

// VECTOR SUBTRACTION

void cudamath::vectorSub(int *a, int *b, int *c, int n)
{
    // Host memory -> Device memory
    int *d_a, *d_b, *d_c;
    multiCudaMalloc(n*sizeof(int), (void **)&d_a, (void **)&d_b, (void **)&d_c);
    cudaCheck( hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice) );
    // Run kernel
    int sm; cudaCheck( hipDeviceGetAttribute(&sm, hipDeviceAttributeMultiprocessorCount, 0) );
    kernels::vectorSub<<<sm, WARPS*32>>>(d_a, d_b, d_c, n);
    // Device memory -> Host memory
    cudaCheck( hipMemcpy(c, d_c, n*sizeof(int), hipMemcpyDeviceToHost) );
    multiCudaFree(d_a, d_b, d_c);
}

void cudamath::vectorInSub(int *a, int *b, int n)
{
    // Host memory -> Device memory
    int *d_a, *d_b;
    multiCudaMalloc(n*sizeof(int), (void **)&d_a, (void **)&d_b);
    cudaCheck( hipMemcpy(d_a, a, n*sizeof(int), hipMemcpyHostToDevice) );
    cudaCheck( hipMemcpy(d_b, b, n*sizeof(int), hipMemcpyHostToDevice) );
    // Run kernel
    int sm; cudaCheck( hipDeviceGetAttribute(&sm, hipDeviceAttributeMultiprocessorCount, 0) );
    kernels::vectorInSub<<<sm, WARPS*32>>>(d_a, d_b, n);
    // Device memory -> Host memory
    cudaCheck( hipMemcpy(a, d_a, n*sizeof(int), hipMemcpyDeviceToHost) );
    multiCudaFree(d_a, d_b);
}

// MACROS

inline void cudaCheck(hipError_t err)
{
    if (err != hipSuccess)
        std::cout << "Cuda error: " << hipGetErrorString(err);
}

inline void multiCudaMalloc(int size, void **a, void **b, void **c)
{
    cudaCheck( hipMalloc(a, size) );
    if (b != NULL) cudaCheck( hipMalloc(b, size) );
    if (c != NULL) cudaCheck( hipMalloc(c, size) );
}

inline void multiCudaFree(void *a, void *b, void *c)
{
    cudaCheck( hipFree(a) );
    if (b != NULL) cudaCheck( hipFree(b) );
    if (c != NULL) cudaCheck( hipFree(c) );
}
